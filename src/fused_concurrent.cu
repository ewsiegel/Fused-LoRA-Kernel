#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <mma.h>  // For nvcuda::wmma
#include <wmma_extension/operators.hpp>

#include "impl.h"  // Ensure this includes the correct Dimensions struct

namespace fused_concurrent {

using ElementInput = half;          // 'half' is equivalent to '__half'
using ElementOutput = half;
using ElementCompute = half;

// Use WMMA namespace
using namespace nvcuda::wmma;

// Define the matrix layouts as type aliases
using LayoutA = col_major;
using LayoutB = col_major;

// Define the tile sizes (must be multiples of 16 for Tensor Cores)
constexpr int WMMA_M = 16;
constexpr int WMMA_N = 16;
constexpr int WMMA_K = 16;

// Kernel using WMMA Tensor Cores
__global__ void fused_concurrent_kernel(
    const ElementInput* __restrict__ W,  // [m x n]
    const ElementInput* __restrict__ x,  // [n x b]
    const ElementInput* __restrict__ B,  // [m x r]
    const ElementInput* __restrict__ A,  // [r x n]
    ElementOutput* __restrict__ Y,       // [m x b]
    int m, int n, int b, int r) {

    // Using WMMA namespace inside the kernel
    using namespace nvcuda::wmma;

    // Coordinates for the output tile
    int row_start = blockIdx.x * WMMA_M;//tile_row * WMMA_M;
    int col_start = blockIdx.y * WMMA_N;//tile_col * WMMA_N;

    // Allocate shared memory
    extern __shared__ char shared_mem[];
    half* shared_u = reinterpret_cast<half*>(shared_mem);
    half* shared_tmp = reinterpret_cast<half*>(shared_mem + r * WMMA_N * sizeof(half));

    fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, ElementCompute> frag_Wx_C;
    fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, ElementCompute> frag_Bu_C;

    // compute Wx
    if (threadIdx.x < 32) {
        fragment<matrix_a, WMMA_M, WMMA_K, WMMA_K, ElementInput, LayoutA> frag_Wx_A;
        fragment<matrix_b, WMMA_K, WMMA_N, WMMA_K, ElementInput, LayoutB> frag_Wx_B;
        fill_fragment(frag_Wx_C, 0.0f);
        for(int k = 0; k < n; k += WMMA_K){
            // compute v = Wx
            // inside this loop to reuse more of x
            int w_row = row_start;
            int w_col = k;
            int x_row = k;
            int x_col = col_start;

            const ElementInput* W_tile_ptr = W + w_col * m + w_row;
            const ElementInput* x_tile_ptr = x + x_col * n + x_row;

            load_matrix_sync(frag_Wx_A, W_tile_ptr, m);
            load_matrix_sync(frag_Wx_B, x_tile_ptr, n);

            mma_sync(frag_Wx_C, frag_Wx_A, frag_Wx_B, frag_Wx_C);
        }
        //store_matrix_sync(shared_tmp, frag_Wx_C, WMMA_M, mem_col_major);
    } else {
        // Fragments for A and x
        fragment<matrix_a, WMMA_M, WMMA_K, WMMA_K, ElementInput, LayoutA> frag_Ax_A;
        fragment<matrix_b, WMMA_K, WMMA_N, WMMA_K, ElementInput, LayoutB> frag_Ax_B;
        fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, ElementCompute> frag_Ax_C;
        // Compute u = Ax
        for(int row = 0; row < r; row += WMMA_M){
            fill_fragment(frag_Ax_C, 0.0f);
            for(int k = 0; k < n; k += WMMA_K){
                int a_row = row;
                int a_col = k;
                int x_row = k;
                int x_col = col_start;

                const ElementInput* A_tile_ptr = A + a_col * r + a_row;
                const ElementInput* x_tile_ptr = x + x_col * n + x_row;

                load_matrix_sync(frag_Ax_A, A_tile_ptr, r);
                load_matrix_sync(frag_Ax_B, x_tile_ptr, n);

                mma_sync(frag_Ax_C, frag_Ax_A, frag_Ax_B, frag_Ax_C);
            }
            store_matrix_sync(shared_u + row, frag_Ax_C, r, mem_col_major);
        }

        // Compute v = Bu
        fragment<matrix_a, WMMA_M, WMMA_K, WMMA_K, ElementInput, LayoutA> frag_Bu_A;
        fragment<matrix_b, WMMA_K, WMMA_N, WMMA_K, ElementInput, LayoutB> frag_Bu_B;
        fill_fragment(frag_Bu_C, 0.0f);

        for(int k = 0; k < r; k += WMMA_K){
            int b_row = row_start;
            int b_col = k;
            int u_row = k;
            int u_col = 0;

            const ElementInput* B_tile_ptr = B + b_col * m + b_row;
            const half* u_tile_ptr = shared_u + u_col * r + u_row;

            load_matrix_sync(frag_Bu_A, B_tile_ptr, m);
            load_matrix_sync(frag_Bu_B, u_tile_ptr, r);

            mma_sync(frag_Bu_C, frag_Bu_A, frag_Bu_B, frag_Bu_C);
        }

        //TODO write frag_Bu_C to shmem
        store_matrix_sync(shared_tmp, frag_Bu_C, WMMA_M, mem_col_major);
    }

    __syncthreads();

    if (threadIdx.x < 32) {
    //if (threadIdx.x >= 32) {
        fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, ElementCompute> frag_Bu_C;
        load_matrix_sync(frag_Bu_C, shared_tmp, WMMA_M, mem_col_major);
        //fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, ElementCompute> frag_Wx_C;
        //load_matrix_sync(frag_Wx_C, shared_tmp, WMMA_M, mem_col_major);

        // accumulate
        frag_Wx_C = frag_Wx_C + frag_Bu_C;

        // Store the result to Y manually, converting from float to half
        ElementOutput* Y_tile_ptr = Y + col_start * m + row_start;
        store_matrix_sync(Y_tile_ptr, frag_Wx_C, m, mem_col_major);
    }
}

void launch_fused_concurrent(
    const __half* d_W, const __half* d_x,
    const __half* d_B, const __half* d_A,
    __half* d_Y, const Dimensions& dims) {

    int m = dims.size_m;
    int n = dims.size_d;
    int b = dims.size_b;
    int r = dims.size_r;

    int threads_per_block = 2*32;
    dim3 gridSize(m / WMMA_M, b / WMMA_N);
    size_t shared_mem_size = r * WMMA_N * sizeof(half) + WMMA_M * WMMA_N * sizeof(half);

    // Check shared memory size
    int max_shared_mem_per_block;
    hipDeviceGetAttribute(&max_shared_mem_per_block, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);

    if(shared_mem_size > static_cast<size_t>(max_shared_mem_per_block)){
        std::cerr << "Shared memory per block exceeds the maximum limit." << std::endl;
        exit(EXIT_FAILURE);
    }

    // Launch the kernel
    fused_concurrent_kernel<<<gridSize, threads_per_block, shared_mem_size>>>(
        d_W,
        d_x,
        d_B,
        d_A,
        d_Y,
        m, n, b, r);

    // Check for errors
    hipError_t err = hipGetLastError();
    if(err != hipSuccess){
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

    // Synchronize to catch errors
    err = hipDeviceSynchronize();
    if(err != hipSuccess){
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

} // namespace fused_concurrent
