#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include "utils.h"
#include "impl.h"

// Namespace for cuBLAS reference implementation
namespace cublas_reference {

// Function that performs the LoRA forward pass using cuBLAS
void launch_cublas_reference(const __half* d_W, const __half* d_x, const __half* d_B, const __half* d_A,
                             __half* d_y, const Dimensions& dims) {
    // Create cuBLAS handle locally
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Device pointers for intermediate results
    __half* d_Wx;
    __half* d_Ax;
    __half* d_BAx;

    size_t size_Wx = dims.size_m * dims.size_b * sizeof(__half);
    size_t size_Ax = dims.size_r * dims.size_b * sizeof(__half);
    size_t size_BAx = dims.size_m * dims.size_b * sizeof(__half);

    hipMalloc(&d_Wx, size_Wx);
    hipMalloc(&d_Ax, size_Ax);
    hipMalloc(&d_BAx, size_BAx);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Step 1: Compute Wx = W * x (size_m x size_b)
    hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        dims.size_m, dims.size_b, dims.size_d,
        &alpha, d_W, HIP_R_16F, dims.size_m, d_x, HIP_R_16F, dims.size_d,
        &beta, d_Wx, HIP_R_16F, dims.size_m,
        HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    // Step 2: Compute Ax = A * x (size_r x size_b)
    hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        dims.size_r, dims.size_b, dims.size_d,
        &alpha, d_A, HIP_R_16F, dims.size_r, d_x, HIP_R_16F, dims.size_d,
        &beta, d_Ax, HIP_R_16F, dims.size_r,
        HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    // Step 3: Compute BAx = B * Ax (size_m x size_b)
    hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        dims.size_m, dims.size_b, dims.size_r,
        &alpha, d_B, HIP_R_16F, dims.size_m, d_Ax, HIP_R_16F, dims.size_r,
        &beta, d_BAx, HIP_R_16F, dims.size_m,
        HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    // Step 4: Add Wx and BAx (y = Wx + BAx)
    int threadsPerBlock = 256;
    int blocksPerGrid = (dims.size_m * dims.size_b + threadsPerBlock - 1) / threadsPerBlock;
    add_matrices_fp16<<<blocksPerGrid, threadsPerBlock>>>(d_Wx, d_BAx, d_y, dims.size_m, dims.size_b);

    // Free intermediate results
    hipFree(d_Wx);
    hipFree(d_Ax);
    hipFree(d_BAx);

    // Destroy cuBLAS handle
    hipblasDestroy(handle);
}

}  // namespace cublas_reference
