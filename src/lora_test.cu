#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <chrono>
#include <cstdlib>
#include <algorithm>  // For std::min
#include <map>
#include <tuple>
#include <vector>
#include <string>

#include "utils.h"
#include "impl.h"

#define EPSILON 1e-6
#define CORRECTNESS true

// Function to initialize an FP32 array with random values and convert to FP16
void initialize_random_fp16(float* array_fp32, __half* array_fp16, int size, float min_value, float max_value, unsigned int seed) {
    std::srand(seed);
    for (int i = 0; i < size; ++i) {
        array_fp32[i] = min_value + static_cast<float>(rand()) / RAND_MAX * (max_value - min_value);
        array_fp16[i] = __float2half(array_fp32[i]);
    }
}

// Struct to store benchmark results
struct BenchmarkResults {
    char const* name;
    std::map<std::tuple<int32_t, int32_t, int32_t, int32_t>, __half*> outputs;
    std::map<std::tuple<int32_t, int32_t, int32_t, int32_t>, double> elapsed_ms;
};

// General benchmark function
template <typename Impl>
BenchmarkResults benchmark(const std::vector<Dimensions>& dimensions_list, int trials) {
    BenchmarkResults results;
    results.name = Impl::name;

    for (const auto& dims : dimensions_list) {
        size_t size_W = dims.size_m * dims.size_d * sizeof(__half);
        size_t size_B = dims.size_m * dims.size_r * sizeof(__half);
        size_t size_A = dims.size_r * dims.size_d * sizeof(__half);
        size_t size_x = dims.size_d * dims.size_b * sizeof(__half);
        size_t size_y = dims.size_m * dims.size_b * sizeof(__half);

        float *h_W_fp32 = (float*)malloc(dims.size_m * dims.size_d * sizeof(float));
        float *h_B_fp32 = (float*)malloc(dims.size_m * dims.size_r * sizeof(float));
        float *h_A_fp32 = (float*)malloc(dims.size_r * dims.size_d * sizeof(float));
        float *h_x_fp32 = (float*)malloc(dims.size_d * dims.size_b * sizeof(float));

        __half *h_W = (__half*)malloc(size_W);
        __half *h_B = (__half*)malloc(size_B);
        __half *h_A = (__half*)malloc(size_A);
        __half *h_x = (__half*)malloc(size_x);
        __half *h_y = (__half*)malloc(size_y);

        unsigned int seed = 42;
        initialize_random_fp16(h_W_fp32, h_W, dims.size_m * dims.size_d, -1000.0f, 1000.0f, seed);
        initialize_random_fp16(h_B_fp32, h_B, dims.size_m * dims.size_r, -1000.0f, 1000.0f, seed);
        initialize_random_fp16(h_A_fp32, h_A, dims.size_r * dims.size_d, -1000.0f, 1000.0f, seed);
        initialize_random_fp16(h_x_fp32, h_x, dims.size_d * dims.size_b, -1000.0f, 1000.0f, seed);

        __half *d_W, *d_B, *d_A, *d_x, *d_y;
        hipMalloc(&d_W, size_W);
        hipMalloc(&d_B, size_B);
        hipMalloc(&d_A, size_A);
        hipMalloc(&d_x, size_x);
        hipMalloc(&d_y, size_y);

        hipMemcpy(d_W, h_W, size_W, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
        hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
        hipMemcpy(d_x, h_x, size_x, hipMemcpyHostToDevice);

        double min_time_ms = 1e9;

        for (int i = 0; i < trials; ++i) {
            auto start = std::chrono::high_resolution_clock::now();
            Impl::run(d_W, d_x, d_B, d_A, d_y, dims);  // Call the templated LoRA implementation
            hipDeviceSynchronize();
            auto end = std::chrono::high_resolution_clock::now();
            std::chrono::duration<double, std::milli> elapsed = end - start;

            if (i > 0) { // Ignore the first trial for warmup
                min_time_ms = std::min(min_time_ms, elapsed.count());
            }
        }

        hipMemcpy(h_y, d_y, size_y, hipMemcpyDeviceToHost);

        // Save results
        auto dim_tuple = std::make_tuple(dims.size_m, dims.size_d, dims.size_b, dims.size_r);
        results.outputs[dim_tuple] = h_y;
        results.elapsed_ms[dim_tuple] = min_time_ms;

        hipFree(d_W);
        hipFree(d_B);
        hipFree(d_A);
        hipFree(d_x);
        hipFree(d_y);
        free(h_W_fp32);
        free(h_B_fp32);
        free(h_A_fp32);
        free(h_x_fp32);
        free(h_W);
        free(h_B);
        free(h_A);
        free(h_x);
        // Do not free h_y as it's stored in results.outputs
    }

    return results;
}

int main() {
    std::vector<Dimensions> dimensions_list = {
        {2048, 2048, 16, 16},
        {2048, 2048, 16, 32},
        {2048, 2048, 16, 64},
        {2048, 2048, 16, 128},
        {2048, 2048, 32, 16},
        {2048, 2048, 32, 32},
        {2048, 2048, 32, 64},
        {2048, 2048, 32, 128},
        {2048, 2048, 64, 16},
        {2048, 2048, 64, 32},
        {2048, 2048, 64, 64},
        {2048, 2048, 64, 128},
        {2048, 2048, 128, 16},
        {2048, 2048, 128, 32},
        {2048, 2048, 128, 64},
        {2048, 2048, 128, 128},
        {2048, 2048, 256, 16},
        {2048, 2048, 256, 32},
        {2048, 2048, 256, 64},
        {2048, 2048, 256, 128},
        {2048, 2048, 512, 16},
        {2048, 2048, 512, 32},
        {2048, 2048, 512, 64},
        {2048, 2048, 512, 128},
        {2048, 2048, 1024, 16},
        {2048, 2048, 1024, 32},
        {2048, 2048, 1024, 64},
        {2048, 2048, 1024, 128},

        {1024, 1024, 16, 16},
        {1024, 1024, 16, 32},
        {1024, 1024, 16, 64},
        {1024, 1024, 16, 128},
        {1024, 1024, 32, 16},
        {1024, 1024, 32, 32},
        {1024, 1024, 32, 64},
        {1024, 1024, 32, 128},
        {1024, 1024, 64, 16},
        {1024, 1024, 64, 32},
        {1024, 1024, 64, 64},
        {1024, 1024, 64, 128},
        {1024, 1024, 128, 16},
        {1024, 1024, 128, 32},
        {1024, 1024, 128, 64},
        {1024, 1024, 128, 128},
        {1024, 1024, 256, 16},
        {1024, 1024, 256, 32},
        {1024, 1024, 256, 64},
        {1024, 1024, 256, 128},
        {1024, 1024, 512, 16},
        {1024, 1024, 512, 32},
        {1024, 1024, 512, 64},
        {1024, 1024, 512, 128},
        {1024, 1024, 1024, 16},
        {1024, 1024, 1024, 32},
        {1024, 1024, 1024, 64},
        {1024, 1024, 1024, 128},

        {512, 512, 16, 16},
        {512, 512, 16, 32},
        {512, 512, 16, 64},
        {512, 512, 16, 128},
        {512, 512, 32, 16},
        {512, 512, 32, 32},
        {512, 512, 32, 64},
        {512, 512, 32, 128},
        {512, 512, 64, 16},
        {512, 512, 64, 32},
        {512, 512, 64, 64},
        {512, 512, 64, 128},
        {512, 512, 128, 16},
        {512, 512, 128, 32},
        {512, 512, 128, 64},
        {512, 512, 128, 128},
        {512, 512, 256, 16},
        {512, 512, 256, 32},
        {512, 512, 256, 64},
        {512, 512, 256, 128},
        {512, 512, 512, 16},
        {512, 512, 512, 32},
        {512, 512, 512, 64},
        {512, 512, 512, 128},
        {512, 512, 1024, 16},
        {512, 512, 1024, 32},
        {512, 512, 1024, 64},
        {512, 512, 1024, 128},

        {256, 256, 16, 16},
        {256, 256, 16, 32},
        {256, 256, 16, 64},
        {256, 256, 16, 128},
        {256, 256, 32, 16},
        {256, 256, 32, 32},
        {256, 256, 32, 64},
        {256, 256, 32, 128},
        {256, 256, 64, 16},
        {256, 256, 64, 32},
        {256, 256, 64, 64},
        {256, 256, 64, 128},
        {256, 256, 128, 16},
        {256, 256, 128, 32},
        {256, 256, 128, 64},
        {256, 256, 128, 128},
        {256, 256, 256, 16},
        {256, 256, 256, 32},
        {256, 256, 256, 64},
        {256, 256, 256, 128},
        {256, 256, 512, 16},
        {256, 256, 512, 32},
        {256, 256, 512, 64},
        {256, 256, 512, 128},
        {256, 256, 1024, 16},
        {256, 256, 1024, 32},
        {256, 256, 1024, 64},
        {256, 256, 1024, 128},

        {128, 128, 16, 16},
        {128, 128, 16, 32},
        {128, 128, 16, 64},
        {128, 128, 16, 128},
        {128, 128, 32, 16},
        {128, 128, 32, 32},
        {128, 128, 32, 64},
        {128, 128, 32, 128},
        {128, 128, 64, 16},
        {128, 128, 64, 32},
        {128, 128, 64, 64},
        {128, 128, 64, 128},
        {128, 128, 128, 16},
        {128, 128, 128, 32},
        {128, 128, 128, 64},
        {128, 128, 128, 128},
        {128, 128, 256, 16},
        {128, 128, 256, 32},
        {128, 128, 256, 64},
        {128, 128, 256, 128},
        {128, 128, 512, 16},
        {128, 128, 512, 32},
        {128, 128, 512, 64},
        {128, 128, 512, 128},
        {128, 128, 1024, 16},
        {128, 128, 1024, 32},
        {128, 128, 1024, 64},
        {128, 128, 1024, 128},
    };
    std::vector<BenchmarkResults> results;

    // Add benchmarks for various implementations
    results.push_back(benchmark<CublasReference>(dimensions_list, 5));
    results.push_back(benchmark<FusedSequential>(dimensions_list, 5));
    // Add other implementations here, e.g., results.push_back(benchmark<OtherImpl>(dimensions_list, 5));

    // Reference output
    const BenchmarkResults& reference_results = results[0];

    // Print results
    for (size_t impl_index = 0; impl_index < results.size(); ++impl_index) {
        const auto& result = results[impl_index];
        std::cout << "Implementation: " << result.name << "\n";
        printf("  %-6s  %-6s  %-6s  %-6s  %-12s  %-14s\n", "size_m", "size_d", "size_b", "size_r", "elapsed_ms", "tflop_per_sec");
        printf("  %-6s  %-6s  %-6s  %-6s  %-12s  %-14s\n", "------", "------", "------", "------", "-----------", "--------------");

        for (const auto& [dims, elapsed_ms] : result.elapsed_ms) {
            auto [size_m, size_d, size_b, size_r] = dims;
            double tflop = 2.0 * size_m * size_d * size_b * 1e-12;
            double tflop_per_sec = tflop / (elapsed_ms * 1e-3);

            double ref_elapsed_ms = reference_results.elapsed_ms.at(dims);
            double speedup = ref_elapsed_ms / elapsed_ms;

            printf("  %-6d  %-6d  %-6d  %-6d  %.2f (%.2fx speedup) %-14.2f\n",
                   size_m, size_d, size_b, size_r, elapsed_ms, speedup, tflop_per_sec);

#if CORRECTNESS
            if (impl_index > 0) { // Skip the reference implementation itself
                const __half* ref_output = reference_results.outputs.at(dims);
                const __half* impl_output = result.outputs.at(dims);

                // Compare outputs
                bool is_correct = true;
                size_t total_elements = size_m * size_b;
                for (size_t i = 0; i < total_elements; ++i) {
                    float ref_val = __half2float(ref_output[i]);
                    float impl_val = __half2float(impl_output[i]);
                    if (std::abs(ref_val - impl_val) > EPSILON) {
                        is_correct = false;
                        break;
                    }
                }

                // Print warning if incorrect
                if (!is_correct) {
                    std::cout << "WARNING: Output mismatch for dimensions ("
                              << size_m << ", " << size_d << ", " << size_b << ", " << size_r
                              << ") in implementation " << result.name << "\n";
                }
            }
#endif
        }

        std::cout << std::endl;
    }

    // Free the stored outputs
    for (auto& result : results) {
        for (auto& [dims, output] : result.outputs) {
            free(output);
        }
    }

    return 0;
}

