#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <wmma_extension/operators.hpp>
#include <hip/hip_cooperative_groups.h>

#include "impl.h"


namespace cg = cooperative_groups;

namespace fused_concurrent_ax {

using ElementInput = half;
using ElementOutput = half;
using ElementCompute = half;

using namespace nvcuda::wmma;

using LayoutA = col_major;
using LayoutB = col_major;

constexpr int WMMA_M = 16;
constexpr int WMMA_N = 16;
constexpr int WMMA_K = 16;

__global__ void fused_concurrent_ax_kernel(
    const ElementInput* __restrict__ W,  // [m x n]
    const ElementInput* __restrict__ x,  // [n x b]
    const ElementInput* __restrict__ B,  // [m x r]
    const ElementInput* __restrict__ A,  // [r x n]
    ElementOutput* __restrict__ Y,       // [m x b]
    int m, int n, int b, int r,
    half *shared_tmp,
    const int LORA_BLOCKS_R,
    const int LORA_BLOCKS_B) {

    using namespace nvcuda::wmma;

    cg::grid_group grid = cg::this_grid();

    int row_start = (blockIdx.x / (b/WMMA_N)) * WMMA_M;
    int col_start = (blockIdx.x % (b/WMMA_N)) * WMMA_N;

    fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, ElementCompute> frag_Wx_C;

    int lora_block_start = gridDim.x - LORA_BLOCKS_R*LORA_BLOCKS_B;

    // compute Wx
    if (blockIdx.x < lora_block_start) {
        fragment<matrix_a, WMMA_M, WMMA_K, WMMA_K, ElementInput, LayoutA> frag_Wx_A;
        fragment<matrix_b, WMMA_K, WMMA_N, WMMA_K, ElementInput, LayoutB> frag_Wx_B;
        fill_fragment(frag_Wx_C, 0.0f);
        for(int k = 0; k < n; k += WMMA_K){
            // compute v = Wx
            int w_row = row_start;
            int w_col = k;
            int x_row = k;
            int x_col = col_start;

            const ElementInput* W_tile_ptr = W + w_col * m + w_row;
            const ElementInput* x_tile_ptr = x + x_col * n + x_row;

            load_matrix_sync(frag_Wx_A, W_tile_ptr, m);
            load_matrix_sync(frag_Wx_B, x_tile_ptr, n);

            mma_sync(frag_Wx_C, frag_Wx_A, frag_Wx_B, frag_Wx_C);
        }
    } else {
        int splits_b = max(16, b / LORA_BLOCKS_B);
        int start_b = ((blockIdx.x - lora_block_start) / LORA_BLOCKS_R) * splits_b;
        int end_b = start_b + splits_b;
        int stop_b = min(b, end_b);

        int splits_r = max(16, r / LORA_BLOCKS_R);
        int start_r = ((blockIdx.x - lora_block_start) % LORA_BLOCKS_R) * splits_r;
        int end_r = start_r + splits_r;
        int stop_r = min(r, end_r);

        for (int col = start_b; col < stop_b; col+=WMMA_N) {
            // Fragments for A and x
            fragment<matrix_a, WMMA_M, WMMA_K, WMMA_K, ElementInput, LayoutA> frag_Ax_A;
            fragment<matrix_b, WMMA_K, WMMA_N, WMMA_K, ElementInput, LayoutB> frag_Ax_B;
            fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, ElementCompute> frag_Ax_C;
            // Compute u = Ax
            for(int row = 0; row < r; row += WMMA_M){
                fill_fragment(frag_Ax_C, 0.0f);
                for(int k = 0; k < n; k += WMMA_K){
                    int a_row = row;
                    int a_col = k;
                    int x_row = k;
                    int x_col = col;

                    const ElementInput* A_tile_ptr = A + a_col * r + a_row;
                    const ElementInput* x_tile_ptr = x + x_col * n + x_row;

                    load_matrix_sync(frag_Ax_A, A_tile_ptr, r);
                    load_matrix_sync(frag_Ax_B, x_tile_ptr, n);

                    mma_sync(frag_Ax_C, frag_Ax_A, frag_Ax_B, frag_Ax_C);
                }
                store_matrix_sync(shared_tmp + col * r + row, frag_Ax_C, r, mem_col_major);
            }
        }
    }

    grid.sync();

    if (blockIdx.x < lora_block_start) {
        //load from rxb u intermediate
        //and do Y = v + Bu
        //for output Y mxb, m = row_start, b = col_start
        //B is mxr
        //so for (m, b) output tile, we need row m of B and column b of u
        fragment<matrix_a, WMMA_M, WMMA_K, WMMA_K, ElementInput, LayoutA> frag_Bu_A;
        fragment<matrix_b, WMMA_K, WMMA_N, WMMA_K, ElementInput, LayoutB> frag_Bu_B;
        fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, ElementCompute> frag_Bu_C;
        fill_fragment(frag_Bu_C, 0.0f);
        for(int k = 0; k < r; k += WMMA_K){
            int b_row = row_start;
            int b_col = k;
            int u_row = k;
            int u_col = col_start;

            const ElementInput* B_tile_ptr = B + b_col * m + b_row;
            const half* u_tile_ptr = shared_tmp + u_col * r + u_row;

            load_matrix_sync(frag_Bu_A, B_tile_ptr, m);
            load_matrix_sync(frag_Bu_B, u_tile_ptr, r);

            mma_sync(frag_Bu_C, frag_Bu_A, frag_Bu_B, frag_Bu_C);
        }

        // accumulate
        frag_Wx_C = frag_Wx_C + frag_Bu_C;

        // Store the result to Y manually, converting from float to half
        ElementOutput* Y_tile_ptr = Y + col_start * m + row_start;
        store_matrix_sync(Y_tile_ptr, frag_Wx_C, m, mem_col_major);
    }
}

void launch_fused_concurrent_ax(
    const __half* d_W, const __half* d_x,
    const __half* d_B, const __half* d_A,
    __half* d_Y, const Dimensions& dims, __half* d_tmp) {

    int m = dims.size_m;
    int n = dims.size_d;
    int b = dims.size_b;
    int r = dims.size_r;

    int threads_per_block = 32;
    int LORA_BLOCKS_R = (r + 15) / 16;
    int LORA_BLOCKS_B = (b + 15) / 16;
    dim3 gridSize(m / WMMA_M * b / WMMA_N + LORA_BLOCKS_R*LORA_BLOCKS_B);

    void* kernelArgs[] = {
        &d_W,
        &d_x,
        &d_B,
        &d_A,
        &d_Y,
        &m, &n, &b, &r,
        &d_tmp,
        &LORA_BLOCKS_R,
        &LORA_BLOCKS_B
    };

    hipError_t err = hipLaunchCooperativeKernel(
        (void*)fused_concurrent_ax_kernel,
        gridSize,
        threads_per_block,
        kernelArgs
    );

    // Check for errors
    if(err != hipSuccess){
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }

}

} // namespace fused_concurrent_ax
